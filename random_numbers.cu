#include "hip/hip_runtime.h"
#include <iostream>
#include "randomgen.h"

using namespace std;

__global__ void Kernel(unsigned int* S, double* W){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	Randomgen obj(S[4*i],S[4*i+1],S[4*i+2],S[4*i+3]);
	W[i]=obj.Rand();
}

int main(){
	srand(17);
	unsigned int *_S;
	double *_W;
	int dim_vec=1024;

	unsigned int *S = new unsigned int[4*dim_vec];
	double *W = new double[dim_vec];

	size_t sizeS = 4*dim_vec * sizeof(unsigned int);
	size_t sizeW = dim_vec * sizeof(double);

	hipMalloc((void**)& _S,sizeS);
	hipMalloc((void**)& _W,sizeW);

	for(int i=0; i<4*dim_vec; i++){
		S[i]=rand()+128;
	}

	hipMemcpy(_S, S, sizeS, hipMemcpyHostToDevice);

	int blockSize=512;
	int gridSize = (dim_vec + blockSize - 1) / blockSize;

	Kernel<<<gridSize, blockSize>>>(_S, _W);

	hipMemcpy(W, _W, sizeW, hipMemcpyDeviceToHost);

	for(int i=0; i<10; i++){
		cout<<W[i]<<endl;
	}

	hipFree(_S);
	hipFree(_W);

	delete[] S;
	delete[] W;

    return 0;
}
